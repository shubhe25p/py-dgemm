#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>


__global__ void test_coleasce(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // if statement is necessary to make things work under tile quantization
  printf("Block DIM X: %d, Block DIM Y: %d, Block IDX X: %d, Block IDX Y: %d, Thread IDX X: %d, Thread IDX Y: %d, X: %d, Y: %d\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, x, y);
}

void run_test_coleasce(int M, int N, int K, float alpha, float *A, float *B,
                     float beta, float *C) {
  
  test_coleasce<<<1, 8>>>(M, N, K, alpha, A, B, beta, C);
}

int main(int argc, char **argv) {

  // get environment variable for device
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }

  printf("Running kernel %d on device %d.\n", 0, deviceIdx);
  run_test_coleasce(m, n, k, alpha, dA, dB, beta, dC);
  return 0;
};